﻿#include <cstdio>
#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>


#include <algorithm>
#include <random>


__global__ void addKernel(int* A, int* B, int* C, int NumElements)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < NumElements)
	{
		C[i] = A[i] + B[i];
	}
}

int main()
{

	std::random_device rd;
	std::default_random_engine generator(rd());
	std::uniform_real_distribution<float> distribution(0,10000);

	const int N = 1024 * 1024;
	const int size = N * sizeof(int);
	std::vector<int> h_A(N), h_B(N), h_C(N);
	for (int i = 0; i < N; i++)
	{
		h_A[i] = distribution(generator);
		h_B[i] = distribution(generator);
		h_C[i] = h_A[i] + h_B[i];
	}
	int* d_A, * d_B, * d_C;
	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);
	hipMemset(d_C, 0, size);

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	addKernel <<<numBlocks, blockSize >>> (d_A, d_B, d_C, N);

	std::vector<int> h_C_result(N);
	hipMemcpy(h_C_result.data(), d_C, size, hipMemcpyDeviceToHost);
	if (!std::equal(h_C.begin(), h_C.end(), h_C_result.begin()))
	{
		std::cerr << "Error: Results do not match!" << std::endl;
	}
	else
	{
		std::cout << "Success: Results match!" << std::endl;
	}

   return 0;
}

